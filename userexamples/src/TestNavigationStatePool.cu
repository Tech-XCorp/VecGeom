#include "hip/hip_runtime.h"
#include "navigation/NavigationState.h"
#include "base/Global.h"
#include "management/CudaManager.h"
#include "navigation/SimpleNavigator.h"
#include "backend/cuda/Backend.h"

#include <stdio.h>

__global__ 
void ProcessNavStates( void* gpu_ptr /* a pointer to buffer of navigation states */, int depth, int n )
{
  const int i = vecgeom::cuda::ThreadIndex();
  if( i >= n ) return;

  if(i==0){
	printf("SIZEOF NAVSTATE ON THE GPU %d\n", sizeof(vecgeom::cuda::NavigationState));
  }

  // get the navigationstate for this thread/lane
  vecgeom::cuda::NavigationState * state = reinterpret_cast<vecgeom::cuda::NavigationState*>( gpu_ptr +
        vecgeom::cuda::NavigationState::SizeOf(depth)*i ); 

  // actually do something to the states; here just popping off the top volume
  state->Pop();

   // state->SetPathPointer( oldpathpointer );
}

void LaunchNavigationKernel( void* gpu_ptr, int depth, int n )
{
  vecgeom::cuda::LaunchParameters launch =
      vecgeom::cuda::LaunchParameters(n);

  ProcessNavStates<<< launch.grid_size , launch.block_size >>>( gpu_ptr, depth, n );
}


