#include "hip/hip_runtime.h"
#include "VecGeom/base/Global.h"
#include "VecGeom/navigation/NavigationState.h"
#include "VecGeom/navigation/NavStatePool.h"
#include "VecGeom/management/CudaManager.h"
#include "VecGeom/navigation/GlobalLocator.h"
#include "VecGeom/backend/cuda/Backend.h"

#include <stdio.h>

__global__ void ProcessNavStates(void *gpu_ptr /* a pointer to buffer of navigation states */, int depth, int n)
{
  using vecgeom::cuda::NavigationState;
  using vecgeom::cuda::NavStatePool;

  const int i = vecgeom::cuda::ThreadIndex();
  if (i >= n) return;

  if (i == 0) {
    printf("*** Size of NavigationState on the GPU: %ld bytes (SizeOf=%ld) at gpu_ptr=%p\n",
           sizeof(vecgeom::cuda::NavigationState), NavigationState::SizeOf(depth), gpu_ptr);
    // dump memory from GPU side
    NavigationState *dumper = reinterpret_cast<NavigationState *>(gpu_ptr);
    dumper->Dump();
  }

  // // // get the navigationstate for this thread/lane
  // NavigationState *states  = reinterpret_cast<NavigationState*>(gpu_ptr);
  // NavigationState *stateA = &(states[i]);
  // // Alternative: forcing size=160 to get to next state
  // NavigationState *state = reinterpret_cast<NavigationState*>(gpu_ptr+i*NavigationState::SizeOf(depth));
  // printf("Alternative state addresses: stateA=%p  |  state=%p\n", stateA, state);

  // get the navigationstate for this thread/lane
  // Warning: arithmetic on pointer to void or function type.
  vecgeom::cuda::NavigationState *state = reinterpret_cast<vecgeom::cuda::NavigationState *>(
      (char *)gpu_ptr + vecgeom::cuda::NavigationState::SizeOf(depth) * i);

  // actually do something to the states; here just popping off the top volume
  printf("From GPU: ");
  state->Print();

  // state->Pop();
  // state->SetPathPointer( oldpathpointer );
}

void LaunchNavigationKernel(void *gpu_ptr, int depth, int n)
{
  vecgeom::cuda::LaunchParameters launch = vecgeom::cuda::LaunchParameters(n);

  int gsize = launch.grid_size.x;
  int bsize = launch.block_size.x;
  printf("Launching GPU kernels: ProcessNavStates<<<%i,%i>>>...\n", gsize, bsize);
  ProcessNavStates<<<launch.grid_size, launch.block_size>>>(gpu_ptr, depth, n);
  printf("Returning from LaunchNavigationKernel now.\n");
}
