#include "hip/hip_runtime.h"
#include <cassert>
#include "testclass.h"

__global__
void do_work_kernel(int *vector) {
  vector[threadIdx.x] *= 10;
}

void gpu::TestClass::DoWork() {
  int *vector_gpu;
  hipMalloc(&vector_gpu, 10*sizeof(int));
  hipMemcpy(vector_gpu, vector, 10*sizeof(int), hipMemcpyHostToDevice);
  do_work_kernel<<<1, 10>>>(vector_gpu);
  hipMemcpy(vector, vector_gpu, 10*sizeof(int), hipMemcpyDeviceToHost);
  assert(hipGetLastError() == hipSuccess);
}