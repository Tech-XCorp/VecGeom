#include "hip/hip_runtime.h"
/// \file TestNavIndex.cu
/// \author Andrei Gheata (andrei.gheata@cern.ch)

#include <VecGeom/base/Transformation3D.h>
#include <VecGeom/management/GeoManager.h>
#include <VecGeom/management/CudaManager.h>
#include <VecGeom/navigation/NavigationState.h>
#include <VecGeom/volumes/PlacedVolume.h>
#include <VecGeom/base/Stopwatch.h>

#include <iomanip>
#include <cassert>
#include <cstdio>

using namespace vecgeom;

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
  if (result) {
    fprintf(stderr, "CUDA error = %s at %s:%d\n", hipGetErrorString(result), file, line);
    hipDeviceReset();
    exit(1);
  }
}

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

namespace visitorcuda {

class GlobalToLocalVisitor {
private:
  int                 fError = 0;   ///< error code

public:
  VECCORE_ATT_HOST_DEVICE
  GlobalToLocalVisitor() {}

  VECCORE_ATT_HOST_DEVICE
  int GetError() const { return fError; }

  VECCORE_ATT_HOST_DEVICE
  void apply(NavStatePath *state, NavIndex_t nav_index)
  {
    unsigned char level            = state->GetLevel();
    int dind                       = 0;
    NavIndex_t nav_ind             = 1;
    VPlacedVolume const *pdaughter = nullptr;
    for (int i = 1; i < level + 1; ++i) {
      pdaughter = state->At(i);
      dind      = pdaughter->GetChildId();
      if (dind < 0) {
        fError = 1;
        return;
      }
      nav_ind = NavStateIndex::PushImpl(nav_ind, pdaughter);
    }

    // Check if navigation index matches input
    if (nav_ind != nav_index) {
      fError = 2;
      return;
    }

    // Check if the physical volume is correct
    if (NavStateIndex::TopImpl(nav_ind) != state->Top()) {
      fError = 3;
      return;
    }

    // Check if the current level is valid
    if (level != NavStateIndex::GetLevelImpl(nav_ind)) {
      fError = 4;
      return;
    }

    // Check if mother navigation index is consistent
    if (level > 0 && nav_ind != NavStateIndex::PushImpl(NavStateIndex::PopImpl(nav_ind), pdaughter)) {
      fError = 5;
      return;
    }

    // Check if the number of daughters is correct
    if (NavStateIndex::GetNdaughtersImpl(nav_ind) != state->Top()->GetDaughters().size()) {
      fError = 6;
      return;
    }

    Transformation3D trans, trans_nav_ind;
    state->TopMatrix(trans);
    NavStateIndex::TopMatrixImpl(nav_ind, trans_nav_ind);
    if (!trans.operator==(trans_nav_ind)) {
      fError = 7;
      return;
    }

    // success
    fError = 0;
  }
};

/// Traverses the geometry tree keeping track of the state context (volume path or navigation state)
/// and applies the injected Visitor
template <typename Visitor>
VECCORE_ATT_HOST_DEVICE
int visitAllPlacedVolumesPassNavIndex(VPlacedVolume const *currentvolume, Visitor *visitor, NavStatePath *state,
                                      NavIndex_t nav_ind)
{
  const char *errcodes[] = {"incompatible daughter pointer",
                            "navigation index mismatch",
                            "top placed volume pointer mismatch",
                            "level mismatch",
                            "navigation index inconsistency for Push/Pop",
                            "number of daughters mismatch",
                            "transformation matrix mismatch"
                           };
  if (currentvolume != NULL) {
    state->Push(currentvolume);
    visitor->apply(state, nav_ind);
    auto ierr = visitor->GetError();
    if (ierr) {
      printf("=== EEE === TestNavIndex: %s\n", errcodes[ierr]);
      return ierr;
    }
    for (auto daughter : currentvolume->GetDaughters()) {
      auto nav_ind_d = NavStateIndex::PushImpl(nav_ind, daughter);
      visitAllPlacedVolumesPassNavIndex(daughter, visitor, state, nav_ind_d);
    }
    state->Pop();
  }
  return 0;
}

} // namespace visitorcuda

__global__
void TestNavIndexGPUKernel(vecgeom::cuda::VPlacedVolume const* const gpu_world, vecgeom::cuda::NavStatePath * const state, int *ierr)
{
  using namespace visitorcuda;
  
  state->Clear();
  GlobalToLocalVisitor visitor;

  NavIndex_t nav_ind_top = 1; // The navigation index corresponding to the world

  *ierr = visitAllPlacedVolumesPassNavIndex(gpu_world, &visitor, state, nav_ind_top);
}

int TestNavIndexGPU(vecgeom::cxx::VPlacedVolume const* const world, int maxdepth)
{
  // Load and synchronize the geometry on the GPU
  size_t statesize = NavigationState::SizeOfInstance(maxdepth);

  vecgeom::cxx::CudaManager::Instance().LoadGeometry(world);
  vecgeom::cxx::CudaManager::Instance().Synchronize();
  
  auto gpu_world = vecgeom::cxx::CudaManager::Instance().world_gpu();
  assert(gpu_world && "GPU world volume is a null pointer");

  char *input_buffer = nullptr;
  checkCudaErrors(hipMallocManaged((void **)&input_buffer, statesize));
  auto state = NavStatePath::MakeInstanceAt(maxdepth, (void *)(input_buffer));

  int ierr;
  int *d_ierr;
  hipMalloc(&d_ierr, sizeof(int));

  Stopwatch timer;
  timer.Start();
  TestNavIndexGPUKernel<<<1, 1>>>(gpu_world, state, d_ierr);
  hipMemcpy(&ierr, d_ierr, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_ierr);

  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipFree(input_buffer));
  auto tvalidate = timer.Stop();
  if (!ierr)
    std::cout << "=== Info navigation table validation on GPU took: " << tvalidate << " sec.\n";

  return ierr;
}

