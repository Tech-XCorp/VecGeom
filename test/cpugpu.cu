#include "hip/hip_runtime.h"
#include "cpugpu.h"

#include "backend/cuda/backend.h"
#include "base/soa3d.h"
#include "management/cuda_manager.h"
#include "navigation/simple_navigator.h"
#include "navigation/navigationstate.h"
#include "volumes/placed_volume.h"

using namespace vecgeom_cuda;

__global__
void LocatePointsKernel(
    vecgeom_cuda::VPlacedVolume const *const world,
    SimpleNavigator const *const navigator,
    NavigationState *const paths, const vecgeom_cuda::SOA3D<Precision> points,
    const int n, int *const output) {
  const int i = ThreadIndex();
  if (i >= n) return;
  output[i] = navigator->LocatePoint(world, points[i], paths[i], true)->id();
}

__global__
void LocatePointsInitialize(
    SimpleNavigator *const navigator, NavigationState *const states,
    const int depth) {
  const int i = ThreadIndex();
  new(&states[i]) NavigationState(depth);
  if (i == 0) new(navigator) SimpleNavigator();
}

void LocatePointsGpu(Precision *const x, Precision *const y, Precision *const z,
                     const unsigned size, const int depth, int *const output) {

  vecgeom_cuda::SOA3D<Precision> points(x, y, z, size);
  Precision *const x_gpu = AllocateOnGpu<Precision>(sizeof(Precision)*size);
  Precision *const y_gpu = AllocateOnGpu<Precision>(sizeof(Precision)*size);
  Precision *const z_gpu = AllocateOnGpu<Precision>(sizeof(Precision)*size);
  points.CopyToGpu(x_gpu, y_gpu, z_gpu, size);

  SimpleNavigator *const navigator = AllocateOnGpu<SimpleNavigator>();
  NavigationState *const paths = AllocateOnGpu<NavigationState>(
    size*sizeof(NavigationState)
  );
  LaunchParameters launch(size);
  LocatePointsInitialize<<<launch.grid_size, launch.block_size>>>(
    navigator, paths, depth
  );
  int *const output_gpu = AllocateOnGpu<int>(size*sizeof(int));

  LocatePointsKernel<<<launch.grid_size, launch.block_size>>>(
    CudaManager::Instance().world_gpu(),
    navigator,
    paths,
    points,
    size,
    output_gpu
  );

  CopyFromGpu(output_gpu, output, size*sizeof(int));

  FreeFromGpu(navigator);
  FreeFromGpu(paths);
  FreeFromGpu(output_gpu);
  FreeFromGpu(x_gpu);
  FreeFromGpu(y_gpu);
  FreeFromGpu(z_gpu);
}