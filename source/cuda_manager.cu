#include "hip/hip_runtime.h"
#include "management/cuda_manager.h"

#include <stdio.h>

#include "backend/cuda/backend.h"

namespace vecgeom {

__global__
void CudaManagerPrintGeometryKernel(
    vecgeom_cuda::VPlacedVolume const *const world) {
  printf("Geometry loaded on GPU:\n");
  world->PrintContent();
}

void CudaManagerPrintGeometry(VPlacedVolume const *const world) {
  CudaManagerPrintGeometryKernel<<<1, 1>>>(
    reinterpret_cast<vecgeom_cuda::VPlacedVolume const*>(world)
  );
  CudaAssertError();
  hipDeviceSynchronize();
}

} // End namespace vecgeom