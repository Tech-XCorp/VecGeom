/// \file BVHManager.cu
/// \author Guilherme Amadio

#include "VecGeom/management/BVHManager.h"
#include "VecGeom/backend/cuda/Interface.h"

using vecgeom::cxx::CudaCheckError;

namespace vecgeom {
inline namespace cuda {
void *AllocateDeviceBVHBuffer(size_t n)
{
  BVH *ptr = nullptr;
  CudaCheckError(hipMalloc((void **)&ptr, n * sizeof(BVH)));
  CudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(dBVH), &ptr, sizeof(ptr)));
  CudaCheckError(hipDeviceSynchronize());
  return (void*) ptr;
}

void FreeDeviceBVHBuffer()
{
  void *ptr = nullptr;

  CudaCheckError(hipMemcpyFromSymbol(&ptr, HIP_SYMBOL(dBVH), sizeof(ptr)));

  if (ptr)
    CudaCheckError(hipFree(ptr));
}
} // namespace cuda
} // namespace vecgeom
