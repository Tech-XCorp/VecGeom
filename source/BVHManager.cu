/// \file BVHManager.cu
/// \author Guilherme Amadio

#include "VecGeom/management/BVHManager.h"
#include "VecGeom/backend/cuda/Interface.h"

using vecgeom::cxx::CudaCheckError;

namespace vecgeom {
inline namespace cuda {

static __device__ BVH *dBVH;

void *AllocateDeviceBVHBuffer(size_t n)
{
  BVH *ptr = nullptr;
  CudaCheckError(hipMalloc((void **)&ptr, n * sizeof(BVH)));
  CudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(dBVH), &ptr, sizeof(ptr)));
  CudaCheckError(hipDeviceSynchronize());
  return (void*) ptr;
}

void FreeDeviceBVHBuffer()
{
  void *ptr = nullptr;

  CudaCheckError(hipMemcpyFromSymbol(&ptr, HIP_SYMBOL(dBVH), sizeof(ptr)));

  if (ptr)
    CudaCheckError(hipFree(ptr));
}

__device__
BVH *GetDeviceBVH(int id)
{
  return &dBVH[id];
}

} // namespace cuda
} // namespace vecgeom
