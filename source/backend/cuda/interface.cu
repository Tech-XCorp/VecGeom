/**
 * @file interface.cu
 * @author Johannes de Fine Licht (johannes.definelicht@cern.ch)
 */

#include <cassert>
#include <iostream>
#include <stdio.h>
 
#include "backend/cuda/interface.h"

#include "base/stopwatch.h"
#include "management/cuda_manager.h"
#include "navigation/simple_navigator.h"
#include "navigation/navigationstate.h"
#include "volumes/placed_volume.h"

namespace vecgeom {

hipError_t CudaCheckError(const hipError_t err) {
  if (err != hipSuccess) {
    std::cout << "CUDA reported error with message: \""
              << hipGetErrorString(err) << "\"\n";
  }
  return err;
}

hipError_t CudaCheckError() {
  return CudaCheckError(hipGetLastError());
}

void CudaAssertError(const hipError_t err) {
  assert(CudaCheckError(err) == hipSuccess);
}

void CudaAssertError() {
  CudaAssertError(hipGetLastError());
}

hipError_t CudaMalloc(void** ptr, unsigned size) {
  return hipMalloc(ptr, size);
}

hipError_t CudaCopyToDevice(void* tgt, void const* src, unsigned size) {
  return hipMemcpy(tgt, src, size, hipMemcpyHostToDevice);
}

hipError_t CudaCopyFromDevice(void* tgt, void const* src, unsigned size) {
  return hipMemcpy(tgt, src, size, hipMemcpyDeviceToHost);
}

hipError_t CudaFree(void* ptr) {
  return hipFree(ptr);
}

} // End namespace vecgeom