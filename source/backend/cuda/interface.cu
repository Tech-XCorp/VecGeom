#include "hip/hip_runtime.h"
/**
 * @file interface.cu
 * @author Johannes de Fine Licht (johannes.definelicht@cern.ch)
 */

#include <cassert>
#include <iostream>
 
#include "backend/cuda/interface.h"
#include "volumes/placed_volume.h"
#include "volumes/logical_volume.h"

namespace vecgeom {

hipError_t CudaCheckError(const hipError_t err) {
  if (err != hipSuccess) {
    std::cout << "CUDA reported error with message: \""
              << hipGetErrorString(err) << "\"\n";
  }
  return err;
}

hipError_t CudaCheckError() {
  return CudaCheckError(hipGetLastError());
}

void CudaAssertError(const hipError_t err) {
  assert(CudaCheckError(err) == hipSuccess);
}

void CudaAssertError() {
  CudaAssertError(hipGetLastError());
}

hipError_t CudaMalloc(void** ptr, unsigned size) {
  return hipMalloc(ptr, size);
}

hipError_t CudaCopyToDevice(void* tgt, void const* src, unsigned size) {
  return hipMemcpy(tgt, src, size, hipMemcpyHostToDevice);
}

hipError_t CudaCopyFromDevice(void* tgt, void const* src, unsigned size) {
  return hipMemcpy(tgt, src, size, hipMemcpyDeviceToHost);
}

hipError_t CudaFree(void* ptr) {
  return hipFree(ptr);
}

// Class specific functions

__global__
void CudaManagerPrintGeometryKernel(
    vecgeom_cuda::VPlacedVolume const *const world) {
  world->logical_volume()->PrintContent();
}

void CudaManagerPrintGeometry(VPlacedVolume const *const world) {
  CudaManagerPrintGeometryKernel<<<1, 1>>>(
    reinterpret_cast<vecgeom_cuda::VPlacedVolume const*>(world)
  );
  CudaAssertError();
  hipDeviceSynchronize();
}

} // End namespace vecgeom