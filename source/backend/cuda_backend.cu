#include "backend/cuda_backend.cuh"

namespace vecgeom {

hipError_t CudaCheckError(const hipError_t err) {
  if (err != hipSuccess) {
    std::cout << "CUDA reported error with message: \""
              << hipGetErrorString(err) << "\"\n";
  }
  return err;
}

hipError_t CudaCheckError() {
  return CudaCheckError(hipGetLastError());
}

void CudaAssertError(const hipError_t err) {
  assert(CudaCheckError(err) == hipSuccess);
}

void CudaAssertError() {
  CudaAssertError(hipGetLastError());
}

void* AllocateOnGpu(const int size) {
  void *ptr;
  CudaAssertError(hipMalloc(&ptr, size));
  return ptr;
}

} // End namespace vecgeom