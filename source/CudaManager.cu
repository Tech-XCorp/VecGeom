#include "hip/hip_runtime.h"
/// \file CudaManager.cu
/// \author Johannes de Fine Licht (johannes.definelicht@cern.ch)

#include "management/CudaManager.h"

#include <stdio.h>

#include "backend/cuda/Backend.h"

namespace vecgeom {
inline namespace cuda {

__global__
void CudaManagerPrintGeometryKernel(
    vecgeom::cuda::VPlacedVolume const *const world) {
  printf("Geometry loaded on GPU:\n");
  world->PrintContent();
}

void CudaManagerPrintGeometry(vecgeom::cuda::VPlacedVolume const *const world) {
  CudaManagerPrintGeometryKernel<<<1, 1>>>(world);
  cxx::CudaAssertError();
  hipDeviceSynchronize();
}

} } // End namespace vecgeom