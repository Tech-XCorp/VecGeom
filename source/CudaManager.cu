#include "hip/hip_runtime.h"
/// \file CudaManager.cu
/// \author Johannes de Fine Licht (johannes.definelicht@cern.ch)

#include "VecGeom/management/CudaManager.h"

#include <stdio.h>

#include "VecGeom/backend/cuda/Backend.h"

namespace vecgeom {

// we put some global data into a separate namespace
// this is done since CUDA does not support static const members in class definitions
namespace globaldevicegeomdata {
static __device__ VPlacedVolume *gCompactPlacedVolBuffer = nullptr;

static __device__ NavIndex_t *gNavIndex = nullptr;

VECCORE_ATT_DEVICE
VPlacedVolume *&GetCompactPlacedVolBuffer()
{
  return gCompactPlacedVolBuffer;
}

VECCORE_ATT_DEVICE
NavIndex_t *&GetNavIndex()
{
  return gNavIndex;
}
}

inline namespace cuda {

__global__ void InitDeviceCompactPlacedVolBufferPtrCudaKernel(void *gpu_ptr)
{
  // gpu_ptr is some pointer on the device that was allocated by some other means
  globaldevicegeomdata::GetCompactPlacedVolBuffer() = (vecgeom::cuda::VPlacedVolume *)gpu_ptr;
}

void InitDeviceCompactPlacedVolBufferPtr(void *gpu_ptr)
{
  InitDeviceCompactPlacedVolBufferPtrCudaKernel<<<1, 1>>>(gpu_ptr);
}

__global__ void InitDeviceNavIndexPtrCudaKernel(void *gpu_ptr)
{
  // gpu_ptr is some pointer on the device that was allocated by some other means
  globaldevicegeomdata::GetNavIndex() = (NavIndex_t *)gpu_ptr;
}

void InitDeviceNavIndexPtr(void *gpu_ptr)
{
  InitDeviceNavIndexPtrCudaKernel<<<1, 1>>>(gpu_ptr);
}

__global__ void CudaManagerPrintGeometryKernel(vecgeom::cuda::VPlacedVolume const *const world)
{
  printf("Geometry loaded on GPU:\n");
  world->PrintContent();
}

void CudaManagerPrintGeometry(vecgeom::cuda::VPlacedVolume const *const world)
{
  CudaManagerPrintGeometryKernel<<<1, 1>>>(world);
  cxx::CudaAssertError();
  hipDeviceSynchronize();
}
}
} // End namespace vecgeom

#ifdef VECCORE_CUDA_SINGLE_OBJ // Cuda single compilation

#include "source/Vector.cpp"
#include "source/SOA3D.cpp"
#include "source/Transformation3D.cpp"

#include "source/LogicalVolume.cpp"
#include "source/PlacedVolume.cpp"
#include "source/UnplacedVolume.cpp"

#include "source/PlacedBox.cpp"
#include "source/UnplacedBox.cpp"
#include "source/SpecializedBox.cpp"

#include "source/PlacedCone.cpp"
#include "source/UnplacedCone.cpp"

#include "source/PlacedTube.cpp"
#include "source/UnplacedTube.cpp"

#include "source/PlacedTorus.cpp"
#include "source/UnplacedTorus.cpp"

#include "source/PlacedTrd.cpp"
#include "source/UnplacedTrd.cpp"

#include "source/PlacedParallelepiped.cpp"
#include "source/UnplacedParallelepiped.cpp"

#include "source/PlacedParaboloid.cpp"
#include "source/UnplacedParaboloid.cpp"

#include "source/PlacedTrapezoid.cpp"
#include "source/UnplacedTrapezoid.cpp"

#include "source/NavigationState.cpp"
#include "source/SimpleNavigator.cpp"

#include "source/UnplacedOrb.cpp"
#include "source/PlacedOrb.cpp"

#include "source/UnplacedSphere.cpp"
#include "source/PlacedSphere.cpp"

#include "source/UnplacedBooleanVolume.cpp"
#include "source/PlacedBooleanVolume.cpp"

#endif
