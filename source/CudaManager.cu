#include "hip/hip_runtime.h"
/// \file CudaManager.cu
/// \author Johannes de Fine Licht (johannes.definelicht@cern.ch)

#include "management/CudaManager.h"

#include <stdio.h>

#include "backend/cuda/Backend.h"

namespace vecgeom {
inline namespace cuda {

__global__ void InitDeviceCompactPlacedVolBufferPtrCudaKernel(void *gpu_ptr) {
  // gpu_ptr is some pointer on the device that was allocated by some other means
  globaldevicegeomdata::GetCompactPlacedVolBuffer() = (vecgeom::cuda::VPlacedVolume *)gpu_ptr;
}

void InitDeviceCompactPlacedVolBufferPtr(void *gpu_ptr) {
  InitDeviceCompactPlacedVolBufferPtrCudaKernel<<<1, 1>>>(gpu_ptr);
}

__global__
void CudaManagerPrintGeometryKernel(
    vecgeom::cuda::VPlacedVolume const *const world) {
  printf("Geometry loaded on GPU:\n");
  world->PrintContent();
}

void CudaManagerPrintGeometry(vecgeom::cuda::VPlacedVolume const *const world) {
  CudaManagerPrintGeometryKernel<<<1, 1>>>(world);
  cxx::CudaAssertError();
  hipDeviceSynchronize();
}

} } // End namespace vecgeom

#ifdef VECGEOM_NVCC_SINGLE_OBJ // Cuda single compilation

#include "source/AOS3D.cpp"
#include "source/Vector.cpp"
#include "source/SOA3D.cpp"
#include "source/Transformation3D.cpp"

#include "source/LogicalVolume.cpp"
#include "source/PlacedVolume.cpp"
#include "source/UnplacedVolume.cpp"

#include "source/PlacedBox.cpp"
#include "source/UnplacedBox.cpp"
#include "source/SpecializedBox.cpp"

#include "source/PlacedCone.cpp"
#include "source/UnplacedCone.cpp"

#include "source/PlacedTube.cpp"
#include "source/UnplacedTube.cpp"

#include "source/PlacedTorus.cpp"
#include "source/UnplacedTorus.cpp"

#include "source/PlacedTrd.cpp"
#include "source/UnplacedTrd.cpp"

#include "source/PlacedParallelepiped.cpp"
#include "source/UnplacedParallelepiped.cpp"

#include "source/PlacedParaboloid.cpp"
#include "source/UnplacedParaboloid.cpp"

#include "source/PlacedTrapezoid.cpp"
#include "source/UnplacedTrapezoid.cpp"


#include "source/NavigationState.cpp"
#include "source/SimpleNavigator.cpp"

#include "source/UnplacedOrb.cpp"
#include "source/PlacedOrb.cpp"

#include "source/UnplacedSphere.cpp"
#include "source/PlacedSphere.cpp"

#include "source/UnplacedBooleanVolume.cpp"
#include "source/PlacedBooleanVolume.cpp"

#endif
