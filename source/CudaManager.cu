#include "hip/hip_runtime.h"
/// \file CudaManager.cu
/// \author Johannes de Fine Licht (johannes.definelicht@cern.ch)

#include "VecGeom/management/CudaManager.h"

#include <stdio.h>

#include "VecGeom/backend/cuda/Backend.h"

namespace vecgeom {

// we put some global data into a separate namespace
// this is done since CUDA does not support static const members in class definitions
namespace globaldevicegeomdata {
static __device__ VPlacedVolume *gCompactPlacedVolBuffer = nullptr;

static __device__ NavIndex_t *gNavIndex = nullptr;

static __device__ int gMaxDepth = 0;

VECCORE_ATT_DEVICE
VPlacedVolume *&GetCompactPlacedVolBuffer()
{
  return gCompactPlacedVolBuffer;
}

VECCORE_ATT_DEVICE
NavIndex_t *&GetNavIndex()
{
  return gNavIndex;
}

VECCORE_ATT_DEVICE
int GetMaxDepth()
{
  return gMaxDepth;
}
}

inline namespace cuda {

__global__ void InitDeviceCompactPlacedVolBufferPtrCudaKernel(void *gpu_ptr)
{
  // gpu_ptr is some pointer on the device that was allocated by some other means
  globaldevicegeomdata::GetCompactPlacedVolBuffer() = (vecgeom::cuda::VPlacedVolume *)gpu_ptr;
}

void InitDeviceCompactPlacedVolBufferPtr(void *gpu_ptr)
{
  InitDeviceCompactPlacedVolBufferPtrCudaKernel<<<1, 1>>>(gpu_ptr);
}

__global__ void InitDeviceNavIndexPtrCudaKernel(void *gpu_ptr, int maxdepth)
{
  // gpu_ptr is some pointer on the device that was allocated by some other means
  globaldevicegeomdata::GetNavIndex() = (NavIndex_t *)gpu_ptr;
  globaldevicegeomdata::gMaxDepth = maxdepth;
}

void InitDeviceNavIndexPtr(void *gpu_ptr, int maxdepth)
{
  InitDeviceNavIndexPtrCudaKernel<<<1, 1>>>(gpu_ptr, maxdepth);
}

__global__ void CudaManagerPrintGeometryKernel(vecgeom::cuda::VPlacedVolume const *const world)
{
  printf("Geometry loaded on GPU:\n");
  world->PrintContent();
}

void CudaManagerPrintGeometry(vecgeom::cuda::VPlacedVolume const *const world)
{
  CudaManagerPrintGeometryKernel<<<1, 1>>>(world);
  cxx::CudaAssertError();
  hipDeviceSynchronize();
}
}
} // End namespace vecgeom
