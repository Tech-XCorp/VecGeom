#include "hip/hip_runtime.h"
/// \file CudaManager.cu
/// \author Johannes de Fine Licht (johannes.definelicht@cern.ch)

#include "VecGeom/management/CudaManager.h"

#include <stdio.h>

#include "VecGeom/backend/cuda/Backend.h"

namespace vecgeom {
inline namespace cuda {

__global__ void InitDeviceCompactPlacedVolBufferPtrCudaKernel(void *gpu_ptr)
{
  // gpu_ptr is some pointer on the device that was allocated by some other means
  globaldevicegeomdata::gCompactPlacedVolBuffer = (vecgeom::cuda::VPlacedVolume *)gpu_ptr;
}

void InitDeviceCompactPlacedVolBufferPtr(void *gpu_ptr)
{
  InitDeviceCompactPlacedVolBufferPtrCudaKernel<<<1, 1>>>(gpu_ptr);
  vecgeom::cxx::CudaAssertError();
}

__global__ void InitDeviceNavIndexPtrCudaKernel(void *gpu_ptr, int maxdepth)
{
  // gpu_ptr is some pointer on the device that was allocated by some other means
  globaldevicegeomdata::gNavIndex = (NavIndex_t *)gpu_ptr;
  globaldevicegeomdata::gMaxDepth = maxdepth;
}

void InitDeviceNavIndexPtr(void *gpu_ptr, int maxdepth)
{
  InitDeviceNavIndexPtrCudaKernel<<<1, 1>>>(gpu_ptr, maxdepth);
}

__global__ void CudaManagerPrintGeometryKernel(vecgeom::cuda::VPlacedVolume const *const world)
{
  printf("Geometry loaded on GPU:\n");
  world->PrintContent();
}

void CudaManagerPrintGeometry(vecgeom::cuda::VPlacedVolume const *const world)
{
  CudaManagerPrintGeometryKernel<<<1, 1>>>(world);
  cxx::CudaAssertError();
  hipDeviceSynchronize();
}
}
} // End namespace vecgeom

#ifdef VECCORE_CUDA_SINGLE_OBJ // Cuda single compilation

#include "source/Vector.cpp"
#include "source/SOA3D.cpp"
#include "source/Transformation3D.cpp"

#include "source/LogicalVolume.cpp"
#include "source/PlacedVolume.cpp"
#include "source/UnplacedVolume.cpp"

#include "source/PlacedBox.cpp"
#include "source/UnplacedBox.cpp"
#include "source/SpecializedBox.cpp"

#include "source/PlacedCone.cpp"
#include "source/UnplacedCone.cpp"

#include "source/PlacedTube.cpp"
#include "source/UnplacedTube.cpp"

#include "source/PlacedTorus.cpp"
#include "source/UnplacedTorus.cpp"

#include "source/PlacedTrd.cpp"
#include "source/UnplacedTrd.cpp"

#include "source/PlacedParallelepiped.cpp"
#include "source/UnplacedParallelepiped.cpp"

#include "source/PlacedParaboloid.cpp"
#include "source/UnplacedParaboloid.cpp"

#include "source/PlacedTrapezoid.cpp"
#include "source/UnplacedTrapezoid.cpp"

#include "source/NavigationState.cpp"
#include "source/SimpleNavigator.cpp"

#include "source/UnplacedOrb.cpp"
#include "source/PlacedOrb.cpp"

#include "source/UnplacedSphere.cpp"
#include "source/PlacedSphere.cpp"

#include "source/UnplacedBooleanVolume.cpp"
#include "source/PlacedBooleanVolume.cpp"

#endif
