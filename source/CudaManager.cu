#include "hip/hip_runtime.h"
/// \file CudaManager.cu
/// \author Johannes de Fine Licht (johannes.definelicht@cern.ch)

#include "management/CudaManager.h"

#include <stdio.h>

#include "backend/cuda/Backend.h"

namespace vecgeom {

__global__
void CudaManagerPrintGeometryKernel(
    vecgeom_cuda::VPlacedVolume const *const world) {
  printf("Geometry loaded on GPU:\n");
  world->PrintContent();
}

void CudaManagerPrintGeometry(vecgeom_cuda::VPlacedVolume const *const world) {
  CudaManagerPrintGeometryKernel<<<1, 1>>>(world);
  CudaAssertError();
  hipDeviceSynchronize();
}

} // End namespace vecgeom