#include "hip/hip_runtime.h"

#include "base/Global.h"

#include "volumes/PlacedVolume.h"
#include "base/SOA3D.h"
#include "navigation/SimpleNavigator.h"
#include "navigation/NavigationState.h"

#ifdef VECGEOM_CUDA
#include "base/Stopwatch.h"
#include "backend/cuda/Backend.h"
#include "management/CudaManager.h"
#include "backend/cuda/Interface.h"
#endif

namespace vecgeom {
inline namespace cuda {

__global__
void NavigationKernel(VPlacedVolume const *const volume,
                   const SOA3D<Precision> positions, const SOA3D<Precision> directions,
                   Precision const * pSteps,  const int n,  Precision *const steps) {

  using NavigationState = vecgeom::cuda::NavigationState;

  // ??? What's the right way to create empty navigation states on the GPU?
  // NavigationState* inState = NavigationState::MakeInstance( CudaManager::Instance().getMaxDepth() );
  // NavigationState* outState = NavigationState::MakeInstance( CudaManager::Instance().getMaxDepth() );

  SimpleNavigator nav;
  // double step;

  unsigned tid = ThreadIndex();
  while (tid < n) {
    steps[tid] = (Precision)tid;

    // ??? Once navigation states are available, uncomment following lines for navigation on the GPU
    // nav.LocatePoint(volume, positions[tid], *inState, true);
    // nav.FindNextBoundaryAndStep(positions[tid], directions[tid], *inState, *outState, pSteps[tid], step);
    // steps[tid] = step;
    tid += ThreadOffset();
  }
}

} // end of namespace cuda

// Should this function be moved to NavigationBenchmarker.cpp?
Precision runNavigationCuda( const cxx::VPlacedVolume *const volume, unsigned npoints,
                             Precision const *const posX, Precision const *const posY, Precision const  *const posZ,
                             Precision const *const dirX, Precision const *const dirY, Precision const *const dirZ,
                             Precision const *const maxSteps,
                             Precision *const propSteps ) {

   // transfer geometry to GPU
   using CudaVolume = cuda::VPlacedVolume const*;
   using CudaSOA3D  = cuda::SOA3D<Precision>;
   using cxx::CudaManager;

   // build a list of GPU volume pointers - needed?

   // copy points to the GPU
   cxx::DevicePtr<Precision> posXGpu; posXGpu.Allocate(npoints);
   cxx::DevicePtr<Precision> posYGpu; posYGpu.Allocate(npoints);
   cxx::DevicePtr<Precision> posZGpu; posZGpu.Allocate(npoints);
   posXGpu.ToDevice(posX, npoints);
   posYGpu.ToDevice(posY, npoints);
   posZGpu.ToDevice(posZ, npoints);
   CudaSOA3D positionGpu = CudaSOA3D(posXGpu, posYGpu, posZGpu, npoints);

   // copy directions to the GPU
   cxx::DevicePtr<Precision> dirXGpu; dirXGpu.Allocate(npoints);
   cxx::DevicePtr<Precision> dirYGpu; dirYGpu.Allocate(npoints);
   cxx::DevicePtr<Precision> dirZGpu; dirZGpu.Allocate(npoints);
   dirXGpu.ToDevice(dirX, npoints);
   dirYGpu.ToDevice(dirY, npoints);
   dirZGpu.ToDevice(dirZ, npoints);
   CudaSOA3D directionGpu = CudaSOA3D(dirXGpu, dirYGpu, dirZGpu, npoints);

   cxx::DevicePtr<Precision> maxStepsGpu;  maxStepsGpu.Allocate(npoints);
   maxStepsGpu.ToDevice(maxSteps, npoints);

   // allocate space for kernel output
   Precision* propStepsGpu = cxx::AllocateOnGpu<Precision>(npoints*sizeof(Precision));

   // launch kernel in GPU
   vecgeom::cuda::LaunchParameters launch(npoints);
   vecgeom::cuda::Stopwatch timer;

   printf("GPU warm-up:  <<<1,32>>>\n");
   vecgeom::cuda::NavigationKernel<<< 1, 32>>>(
     CudaManager::Instance().world_gpu(),
     positionGpu, directionGpu,
     maxStepsGpu, 32, propStepsGpu
     );
   hipDeviceSynchronize();

   printf("GPU configuration:  <<<%i,%i>>>\n", launch.grid_size.x, launch.block_size.x);

   timer.Start();
   vecgeom::cuda::NavigationKernel<<< launch.grid_size, launch.block_size>>>(
       CudaManager::Instance().world_gpu(),
       positionGpu, directionGpu, maxStepsGpu, npoints, propStepsGpu
     );
   hipDeviceSynchronize();
   Precision elapsedCuda = timer.Stop();

   cxx::CopyFromGpu(propStepsGpu, propSteps, npoints*sizeof(Precision));

   cxx::FreeFromGpu(propStepsGpu);
   posXGpu.Deallocate();
   posYGpu.Deallocate();
   posZGpu.Deallocate();
   dirXGpu.Deallocate();
   dirYGpu.Deallocate();
   dirZGpu.Deallocate();

   // compare steps from navigator with the ones above
   std::cout<<"GPU navigation time: "<< 1000.*elapsedCuda <<" ms\n";
   return elapsedCuda;
}

} // global namespace
