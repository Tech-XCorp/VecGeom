#include "hip/hip_runtime.h"

#include "base/Global.h"

#include "volumes/PlacedVolume.h"
#include "base/SOA3D.h"
#include "navigation/SimpleNavigator.h"
#include "navigation/NavigationState.h"
#include "management/CudaManager.h"
#include "backend/cuda/Backend.h"

#ifdef VECGEOM_CUDA
#include "base/Stopwatch.h"
#include "backend/cuda/Backend.h"
#include "management/CudaManager.h"
#endif

namespace vecgeom_cuda
{

__global__
void NavKernelTest(
    VPlacedVolume const *const volume,
    const SOA3D<Precision> positions,
    const SOA3D<Precision> directions,
    const int n,
    Precision *const steps) {

  unsigned tid = ThreadIndex(); 
NavigationState old(5), newstate(5);
SimpleNavigator nav;
nav.LocatePoint(volume,positions[tid],old,true);
double step;
nav.FindNextBoundaryAndStep(positions[tid],directions[tid], old, newstate, kInfinity, step);
steps[tid]=step;            
//distance[tid] = volume->SafetyToOut(positions[tid]);

}

} // end of namespace vecgeom_cuda

namespace vecgeom
{

  template <typename Type>
  Type* AllocateAligned(unsigned n) {
    return (Type*) _mm_malloc(n*sizeof(Type), kAlignmentBoundary);
  }

  template <typename Type>
  void FreeAligned(Type *const pmemory) {
    if (pmemory) _mm_free(pmemory);
  }

  void RunNavigationCuda( VPlacedVolume const* volume, unsigned npoints,
                          Precision *const posX, Precision *const posY, Precision *const posZ,
                          Precision *const dirX, Precision *const dirY, Precision *const dirZ,
                          Precision *const distances, Precision *const safeties) {

#ifdef VECGEOM_CUDA
   // transfer geometry to GPU
   typedef vecgeom_cuda::VPlacedVolume const* CudaVolume;
   typedef vecgeom_cuda::SOA3D<Precision> CudaSOA3D;

   CudaManager::Instance().LoadGeometry(volume);
   CudaManager::Instance().Synchronize();
   // std::list<CudaVolume> volumesGpu;
   // for (std::list<VolumePointers>::const_iterator v = fVolumes.begin();
   //      v != fVolumes.end(); ++v) {
   //   volumesGpu.push_back(
   //     reinterpret_cast<CudaVolume>(
   //       CudaManager::Instance().LookupPlaced(v->Specialized())
   //     )
   //   );
   // }

   // copy points to the GPU
   Precision *posXGpu = AllocateOnGpu<Precision>(sizeof(Precision)*npoints);
   Precision *posYGpu = AllocateOnGpu<Precision>(sizeof(Precision)*npoints);
   Precision *posZGpu = AllocateOnGpu<Precision>(sizeof(Precision)*npoints);
   CopyToGpu(posX, posXGpu, npoints*sizeof(Precision));
   CopyToGpu(posY, posYGpu, npoints*sizeof(Precision));
   CopyToGpu(posZ, posZGpu, npoints*sizeof(Precision));
   CudaSOA3D positionGpu = CudaSOA3D(posXGpu, posYGpu, posZGpu, npoints);

   // copy directions to the GPU
   Precision *dirXGpu = AllocateOnGpu<Precision>(sizeof(Precision)*npoints);
   Precision *dirYGpu = AllocateOnGpu<Precision>(sizeof(Precision)*npoints);
   Precision *dirZGpu = AllocateOnGpu<Precision>(sizeof(Precision)*npoints);
   CopyToGpu(dirX, dirXGpu, npoints*sizeof(Precision));
   CopyToGpu(dirY, dirYGpu, npoints*sizeof(Precision));
   CopyToGpu(dirZ, dirZGpu, npoints*sizeof(Precision));
   CudaSOA3D directionGpu = CudaSOA3D(dirXGpu, dirYGpu, dirZGpu, npoints);

   // allocate space for kernel output
   Precision *stepsGpu = AllocateOnGpu<Precision>(sizeof(Precision)*npoints);

   vecgeom_cuda::LaunchParameters launch =
     vecgeom_cuda::LaunchParameters(npoints);

   // launch kernel in GPU
   vecgeom_cuda::Stopwatch timer;
   timer.Start();
   // for (unsigned r = 0; r < fRepetitions; ++r) {
   vecgeom_cuda::NavKernelTest<<< launch.grid_size, launch.block_size>>>(
     *volume, positionGpu, directionGpu, npoints, stepsGpu );
   // }
   Precision elapsedCuda = timer.Stop();

   Precision *const stepsCpu = AllocateAligned<Precision>();
   Precision * steps = (Precision *) _mm_malloc(sizeof(Precision)*np,32);
   CopyFromGpu(stepsGpu, stepsCpu, npoints*sizeof(Precision));

   FreeFromGpu(stepsGpu);
   FreeFromGpu(posXGpu);
   FreeFromGpu(posYGpu);
   FreeFromGpu(posZGpu);
   FreeFromGpu(dirXGpu);
   FreeFromGpu(dirYGpu);
   FreeFromGpu(dirZGpu);

   // compare steps from navigator with the ones above
   std::cout<<"TODO: compare steps from navigator with the previous ones.\n";
#endif  // VECGEOM_CUDA
  }

}
